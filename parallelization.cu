#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <iomanip>
using namespace std;

#define time 86400
#define AU_TO_M 1.496e+11
#define M_TO_AU 1.0/1.496e+11
#define AU_PER_DAY_TO_MS 1.731e+6
#define MS_TO_AU_PER_DAY 1.0/1.731e+6
#define G 6.67430e-11
#define N 9
#define DAY 1000

__global__ void compute_gravitational_force(double *old_pos, double *old_vel, double *mass, double *new_pos, double *new_vel)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    if (i>= N || i==0) return; 

    double ax = 0.0, ay = 0.0, az = 0.0;
    double new_ax = 0.0, new_ay = 0.0, new_az = 0.0;

    
    for (int j=0;j<N;j++) {
        if (i!=j) {
            
            double dx=old_pos[j*3]-old_pos[i*3];
            double dy=old_pos[j*3+1]-old_pos[i*3+1];
            double dz=old_pos[j*3+2]-old_pos[i*3+2];
            double squared_distance=fmax(dx*dx+dy*dy+dz*dz, 1e-9);
            double distance=sqrt(squared_distance);
            double force=(G*mass[j])/(distance*squared_distance);

            ax+=force*dx;
            ay+=force*dy;
            az+=force*dz;
        }
    }

   
    double vx_1=old_vel[i*3]+0.5*ax*time;
    double vy_1=old_vel[i*3+1]+0.5*ay*time;
    double vz_1=old_vel[i*3+2]+0.5*az*time;

    new_pos[i*3]=old_pos[i*3]+vx_1*time;
    new_pos[i*3+1]=old_pos[i*3+1]+vy_1*time;
    new_pos[i*3+2]=old_pos[i*3+2]+vz_1*time;

    for (int j=0;j<N;j++) {
        if (i!= j) {

            double dx=new_pos[j*3]-new_pos[i*3];
            double dy=new_pos[j*3+1]-new_pos[i*3+1];
            double dz=new_pos[j*3+2]-new_pos[i*3+2];
            double squared_distance=fmax(dx*dx+dy*dy+dz*dz,1e-9);
            double distance=sqrt(squared_distance);            
            double force=(G*mass[j])/(distance*squared_distance);

            new_ax+=force*dx;
            new_ay+=force*dy;
            new_az+=force*dz;
        }
    }
    
    new_vel[i*3]=vx_1+0.5*new_ax*time;
    new_vel[i*3+1]=vy_1+0.5*new_ay*time;
    new_vel[i*3+2]=vz_1+0.5*new_az*time;
}


vector<vector<double>> reading_csv(const string &file_name) 
{
    ifstream file(file_name);
    vector<vector<double>> data;
    string line;
    getline(file, line);
    
    while (getline(file,line)) 
    {
        stringstream ss(line);
        vector<double> row;
        string value;
        getline(ss, value, ',');
        while (getline(ss, value, ',')) 
        {
            row.push_back(stod(value)); 
            
        }

        data.push_back(row);
    }
   
    file.close();
    return data;
}


void flatten_data(vector<vector<double>> &planet_data,double *pos, double *vel , double *mass)
{
    for(int i=0;i<planet_data.size();i++){
        mass[i]=planet_data[i][0]; 
        pos[i*3]=planet_data[i][1]*AU_TO_M;  
        pos[i*3+1]=planet_data[i][2]*AU_TO_M; 
        pos[i*3+2]=planet_data[i][3]*AU_TO_M;
        vel[i*3]=planet_data[i][4]*AU_PER_DAY_TO_MS;  
        vel[i*3+1]=planet_data[i][5]*AU_PER_DAY_TO_MS;  
        vel[i*3+2]=planet_data[i][6]*AU_PER_DAY_TO_MS;  
         
    }
}

double* to_cuda_memory(double *host_data,int n){
    double *device_data;
    hipMalloc(&device_data,n*sizeof(double));
    hipMemcpy(device_data,host_data,n*sizeof(double),hipMemcpyHostToDevice);
    return device_data;
}

void save_to_csv(int day, double *position, double *velocity) {
    vector<string> planet_names={
        "Sun", "Mercury", "Venus", "Earth", "Mars", "Jupiter", "Saturn", "Uranus", "Neptune"
    };

    for (int i = 0; i < N; i++) {
        string filename=planet_names[i]+".csv";
        ofstream file;
        file.open(filename, ios::app);        
        if (day == 1) {
            file << "Day,X,Y,Z,VX,VY,VZ\n";
        }

        file<<day<<"," 
             <<position[i*3]*M_TO_AU<<","<<position[i*3+1]*M_TO_AU<<","<<position[i*3+2]*M_TO_AU<<","
             <<velocity[i*3]*MS_TO_AU_PER_DAY<<","<<velocity[i*3+1]*MS_TO_AU_PER_DAY<<","<<velocity[i*3+2]*MS_TO_AU_PER_DAY<<"\n";
        file.close();
    }
}

int main() 
{
    string file_name="solar_system.csv";  
    vector<vector<double>> planet_data = reading_csv(file_name);
    int row=planet_data.size();
    int col=planet_data[0].size();

    double *position=new double[N*3];
    double *velocity=new double[N*3];
    double *mass=new double[N];

    
    flatten_data(planet_data,position,velocity,mass);

    double *position_c=to_cuda_memory(position,N*3);
    double *velocity_c=to_cuda_memory(velocity,N*3);
    double *mass_c=to_cuda_memory(mass,N);

    int block_size=9;
    int grid_size=1;
    double *new_position,*new_velocity;
    hipMalloc(&new_position,N*3*sizeof(double));
    hipMalloc(&new_velocity,N*3*sizeof(double));
       

    for (int day = 1; day <= DAY; day++) {
        compute_gravitational_force<<<grid_size, block_size>>>(position_c, velocity_c, mass_c, new_position, new_velocity);
        hipDeviceSynchronize();

        hipMemcpy(position, new_position, N * 3 * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(velocity, new_velocity, N * 3 * sizeof(double), hipMemcpyDeviceToHost);

        save_to_csv(day, position, velocity);

        swap(position_c, new_position);
        swap(velocity_c, new_velocity);
    }
    
    hipFree(position_c);
    hipFree(velocity_c);
    hipFree(mass_c);
    hipFree(new_position);
    hipFree(new_velocity);
    return 0;
}
